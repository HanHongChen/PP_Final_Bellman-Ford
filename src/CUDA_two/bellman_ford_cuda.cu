#include "hip/hip_runtime.h"
#include <climits>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include "../../common/graph.h"

#define USE_BINARY_GRAPH 1
#define DISTANCE_INFINITY 1000000

const int BLOCK_SIZE = 256;

__global__ void relaxEdges(int n, int *distances, int *mat, bool *updated) {

    int v = blockIdx.x * blockDim.x + threadIdx.x;
    int v_stride = gridDim.x * blockDim.x;

    for(int i = 0 ; i < n ; i++){
    	for(int j = v; j < n; j += v_stride){
    	    int weight = mat[i * n + j];
    	    if(weight != DISTANCE_INFINITY){
    	        int new_dist = distances[i] + weight;
    		if(new_dist < distances[j]){
    		    distances[j] = new_dist;
    		    *updated = true;
    		}
    	    }
    	}
    }
    /*
    while (v < num_nodes) {
        int start_edge = outgoing_starts[v];
        int end_edge = (v == num_nodes - 1) ? num_edges : outgoing_starts[v + 1];
        
	for(int edge_idx = start_edge; edge_idx < end_edge; edge_idx++){
            int u = outgoing_edges[edge_idx];
            int weight = edge_cost[edge_idx];
            
	    if(distances[v] == DISTANCE_INFINITY)
  	        break;

            if (distances[v] + weight < distances[u]) {
                distances[u] = distances[v] + weight;
                *updated = true;
            }
	}
	v += v_stride;
    }*/

}

int bellman_ford_cuda(int num_nodes, int *mat, Graph g, int *dis) {
    int *dev_distances, *dev_mat;
    bool *dev_updated;
    bool updated = true;
    int num_edges = g->num_edges;
    int *distances = g->distances;
    // initialize source to 0
    distances[0] = 0;

    hipMalloc((void**)&dev_mat, num_nodes * num_nodes * sizeof(int));
    hipMalloc((void**)&dev_distances, num_nodes * sizeof(int));
    hipMalloc((void**)&dev_updated, sizeof(bool));

    hipMemcpy(dev_distances, distances, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_mat, mat, num_nodes * num_nodes * sizeof(int), hipMemcpyHostToDevice);

    int iteration = 0;
    
    while (updated && iteration < num_nodes - 1) {
        updated = false;
        hipMemcpy(dev_updated, &updated, sizeof(bool), hipMemcpyHostToDevice);

        relaxEdges<<<(num_edges + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(num_nodes, dev_distances, dev_mat, dev_updated);
        hipDeviceSynchronize();

        hipMemcpy(&updated, dev_updated, sizeof(bool), hipMemcpyDeviceToHost);
        iteration++;
    }
    
    //std::cout<<"iterations:"<<iteration<<std::endl;

    int* distances_result = (int*)malloc(num_nodes * sizeof(int));
    hipMemcpy(distances_result, dev_distances, num_nodes * sizeof(int), hipMemcpyDeviceToHost);

    for (int v = 0; v < num_nodes; v++) {
        distances[v] = distances_result[v];
    }

    free(distances_result);
    hipFree(dev_distances);
    hipFree(dev_updated);
    hipFree(dev_mat);
    
    for(int v=0; v<g->num_nodes; v++){
        int start_edge = g->outgoing_starts[v];
        int end_edge = (v == g->num_nodes - 1)? g->num_edges: g->outgoing_starts[v + 1];

        //for each u, which is outgoing neighbor of v
        for(int edge_idx = start_edge; edge_idx < end_edge; edge_idx++){
            Vertex u = g->outgoing_edges[edge_idx];
            if( g->distances[v] == DISTANCE_INFINITY) // v can't relax ant neighbor
                break;
            if(g->distances[v] + g->edge_cost[edge_idx] < g->distances[u] ){
                return 1;
            }
        }
    }
    return 0;
}

int main(int argc, char** argv) {
    // ----------------- Parsing The Input, and load graph .-------------------------
    // You can check the graph I print on the terminal
    if(argc < 2){
        std::cerr << "Usage: " << argv[0] << " <graph>" << std::endl;
        return 1;
    }
    std::string graph_filename = argv[1];
    Graph g = (Graph)malloc(1*sizeof(graph));
    if (USE_BINARY_GRAPH) {
        g = load_graph_binary(graph_filename.c_str());
    } else {
        g = load_graph(argv[1]);
        printf("storing binary form of graph!\n");
        store_graph_binary(graph_filename.append(".bin").c_str(), g);
        free_graph(g);
        exit(1);
    }
    //print_graph(g);

    // write g to an adjacent matrix
    int *mat;
    int num_nodes = g->num_nodes;
    int num_edges = g->num_edges;
    int *distances;
    distances = (int *) malloc(num_nodes * sizeof(int));
    long long int n = (long long int)num_nodes * num_nodes;
    mat = (int *) malloc(n * sizeof(int));
    
    //printf("num_nodes:%d, n:%lld\n", num_nodes, n); 
    // initialize all elements to INFINITY
    for(long long int i = 0; i < n; i++)
        mat[i] = DISTANCE_INFINITY;

    //printf("success\n");
    // for-loop all nodes for setting edge cost in mat
    for (long long int i=0; i<num_nodes; i++) {
        int start_edge = g->outgoing_starts[i];
        int end_edge = (i == num_nodes-1) ? num_edges : g->outgoing_starts[i+1];
        for (int j=start_edge; j<end_edge; j++) {
            int target = g->outgoing_edges[j];
	    long long int idx = (long long int) i * num_nodes + target;
	    //printf("%lld\n", idx);
            mat[idx] = g->edge_cost[j];
        }
    }
    
    /*
    for(long long int i = 0; i < n; i++){
        printf("%d ", mat[i]);
	if((i + 1) % num_nodes == 0)
	    printf("\n");
    }
    */
    
    //printf("success\n");

    //--------------------- start running "Bellmam Ford"--------------------------
    auto start_time = std::chrono::high_resolution_clock::now();
    
    int exits_negative_cycle = bellman_ford_cuda(num_nodes, mat, g, distances);

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

    //------------------------ Check  result-------------------------------
    if(exits_negative_cycle){
        printf("OMG!!! exist negative cycle!!!!!!!!!!!!!!!\n");
    }else{
        print_distances(g, "");
        /*for(int i=0; i<num_nodes; i++){
            printf("distance[%d] = %d\n", i, distances[i]);
	}*/
    }   
    std::cout << "Bellman Ford Cuda 2: " << duration.count() << " microseconds" << std::endl;
 
    return 0;
}

