#include "hip/hip_runtime.h"
#include <climits>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include "../../common/graph.h"

#define USE_BINARY_GRAPH 1
#define DISTANCE_INFINITY 1000000

const int BLOCK_SIZE = 256;

__global__ void relaxEdges(Vertex *outgoing_edges, int *outgoing_starts, int *edge_cost, int num_nodes, int num_edges, int *distances, bool *updated) {

    int v = blockIdx.x * blockDim.x + threadIdx.x;
    int v_stride = gridDim.x * blockDim.x;

    while (v < num_nodes) {
        int start_edge = outgoing_starts[v];
        int end_edge = (v == num_nodes - 1) ? num_edges : outgoing_starts[v + 1];
        
	for(int edge_idx = start_edge; edge_idx < end_edge; edge_idx++){
            int u = outgoing_edges[edge_idx];
            int weight = edge_cost[edge_idx];
            
	    if(distances[v] == DISTANCE_INFINITY)
  	        break;

            if (distances[v] + weight < distances[u]) {
                distances[u] = distances[v] + weight;
                *updated = true;
            }
	}
	v += v_stride;
    }

}

int bellman_ford_cuda(Graph g) {
    // initialize source to 0
    g->distances[g->source] = 0;
    
    int *dev_outgoing_starts, *dev_edge_cost, *dev_distances;
    Vertex *dev_outgoing_edges;
    bool *dev_updated;
    bool updated = true;
    int *distances = g->distances;
    int num_nodes = g->num_nodes;
    int num_edges = g->num_edges;

    hipMalloc((void**)&dev_outgoing_starts, num_nodes * sizeof(int));
    hipMalloc((void**)&dev_edge_cost, num_edges * sizeof(int));
    hipMalloc((void**)&dev_outgoing_edges, num_edges * sizeof(Vertex));
    hipMalloc((void**)&dev_distances, num_nodes * sizeof(int));
    hipMalloc((void**)&dev_updated, sizeof(bool));

    hipMemcpy(dev_outgoing_starts, g->outgoing_starts, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_edge_cost, g->edge_cost, num_edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_outgoing_edges, g->outgoing_edges, num_edges * sizeof(Vertex), hipMemcpyHostToDevice);
    hipMemcpy(dev_distances, g->distances, num_nodes * sizeof(int), hipMemcpyHostToDevice);

    int iteration = 0;
    
    while (updated && iteration < num_nodes - 1) {
        updated = false;
        hipMemcpy(dev_updated, &updated, sizeof(bool), hipMemcpyHostToDevice);

        relaxEdges<<<(num_edges + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(dev_outgoing_edges, dev_outgoing_starts, dev_edge_cost, num_nodes, num_edges, dev_distances, dev_updated);
        hipDeviceSynchronize();

        hipMemcpy(&updated, dev_updated, sizeof(bool), hipMemcpyDeviceToHost);
        iteration++;
    }
    
    // std::cout<<"iterations:"<<iteration<<std::endl;

    int* distances_result = (int*)malloc(num_nodes * sizeof(int));
    hipMemcpy(distances_result, dev_distances, num_nodes * sizeof(int), hipMemcpyDeviceToHost);

    for (int v = 0; v < num_nodes; v++) {
        distances[v] = distances_result[v];
    }

    free(distances_result);
    hipFree(dev_outgoing_starts);
    hipFree(dev_edge_cost);
    hipFree(dev_outgoing_edges);
    hipFree(dev_distances);
    hipFree(dev_updated);
    
    for(int v=0; v<g->num_nodes; v++){
        int start_edge = g->outgoing_starts[v];
        int end_edge = (v == g->num_nodes - 1)? g->num_edges: g->outgoing_starts[v + 1];

        //for each u, which is outgoing neighbor of v
        for(int edge_idx = start_edge; edge_idx < end_edge; edge_idx++){
            Vertex u = g->outgoing_edges[edge_idx];
            if( g->distances[v] == DISTANCE_INFINITY) // v can't relax ant neighbor
                break;
            if(g->distances[v] + g->edge_cost[edge_idx] < g->distances[u] ){
                return 1;
            }
        }
    }
    return 0;
}

int main(int argc, char** argv) {
    // ----------------- Parsing The Input, and load graph .-------------------------
    // You can check the graph I print on the terminal
    if(argc < 2){
        std::cerr << "Usage: " << argv[0] << " <graph>" << std::endl;
        return 1;
    }
    std::string graph_filename = argv[1];
    Graph g = (Graph)malloc(1*sizeof(graph));
    if (USE_BINARY_GRAPH) {
        g = load_graph_binary(graph_filename.c_str());
    } else {
        g = load_graph(argv[1]);
        printf("storing binary form of graph!\n");
        store_graph_binary(graph_filename.append(".bin").c_str(), g);
        free_graph(g);
        exit(1);
    }
    print_graph(g);
    //--------------------- start running "Bellmam Ford"--------------------------
    auto start_time = std::chrono::high_resolution_clock::now();
    
    int exits_negative_cycle = bellman_ford_cuda(g);

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

    //------------------------ Check  result-------------------------------
    if(exits_negative_cycle){
        printf("OMG!!! exist negative cycle!!!!!!!!!!!!!!!\n");
    }else{
        print_distances(g, "");
    }   
    std::cout << "Bellman Ford Serial: " << duration.count() << " microseconds" << std::endl;
 
    return 0;
}

